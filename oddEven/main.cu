#include<iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void exchangeMin(int* arr, int start){
    int tID = blockDim.x*blockIdx.x + threadIdx.x;
    if (arr[start + tID*2] <= arr[start + tID*2 + 1]){
        return;
    }

    int temp = arr[start + tID*2];
    arr[start + tID*2] = arr[start + tID*2 + 1];
    arr[start + tID*2 + 1] = temp;
}

void cudaOddEvenSort(int* arr, int length){
    for (int i = 0; i<length; i++){
        if (0 == i%2){
            exchangeMin<<<1, length/2, 0, 0>>>(arr, 0);
        }
        else{
            exchangeMin<<<1, (length-1)/2, 0, 0>>>(arr, 1);
        }
    }
}

int main(){
    int length = 9;
    int arr[9]={1, 4, 7, 3, 5, 0, 15, 9, 12};

    int* cudaArr;
    hipError_t err = hipMalloc(&cudaArr, length*sizeof(int));
    if (hipSuccess != err){
        cout<<"hipMalloc err "<<err<<endl;
        return err;
    }

    err = hipMemcpy(cudaArr, arr, length*sizeof(int), hipMemcpyHostToDevice);
    if (hipSuccess != err){
        cout<<"hipMemcpy err "<<err<<endl;
        return err;
    }

    cudaOddEvenSort(cudaArr, length);

    err = hipMemcpy(arr, cudaArr, length*sizeof(int), hipMemcpyDeviceToHost);
    if (hipSuccess != err){
        cout<<"hipMemcpy err "<<err<<endl;
        return err;
    }

    for (int i = 0; i<length; i++){
        cout<<arr[i]<<endl;
    }

    hipFree(cudaArr);

    return 0;
}