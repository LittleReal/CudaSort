#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>
#include<device_function.h>

#define INT_MAX 2147483647

#define Num 1024

using namespace std;
  
  
__device__ void swap(int &a, int &b){
    int t = a;
    a = b;
    b = t;
}

 
//for > 1024
__global__ void bigBinoticSort(int *arr, int len, int lenMax) {
	unsigned tid = threadIdx.x;
    if (tid >= lenMax) return;
    
	unsigned iter = blockDim.x;
	for (unsigned i = tid; i < lenMax; i += iter) {
		if (i >= len) {
			arr[i] = INT_MAX;
		}
	}
	
	__syncthreads();
	
	int count = 0;
	for (unsigned i = 2; i <= lenMax; i<<=2) {
		for (unsigned j = i >> 1; j > 0; j >>= 1) {
			for (unsigned k = tid; k < lenMax; k += iter) {
                unsigned swapIdx = k ^ j;
                
                if(swapIdx > k){
                    if(((k & i) == 0)){
                        if(arr[k] > arr[swapIdx]){
                            swap(arr[k], arr[swapIdx]);
                        }
                    }
                    else{
                        if(arr[k] < arr[swapIdx]){
                            swap(arr[k], arr[swapIdx]);
                        }
                    }
                }
			}
			__syncthreads();
		}
	}
}


// for <=1024
__global__ void littleBinoticSort(int* arr,int num, int numMax){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid >= num) arr[tid] = INT_MAX;

    __syncthreads();

    for(unsigned int i=2; i<=numMax; i<<=1){
        for(unsigned int j=i>>1; j>0; j>>=1){
            unsigned int swapIdx = tid ^ j;

            if(swapIdx > tid){
                if((tid & i)==0){
                    if(arr[tid] > arr[swapIdx]){
                        swap(arr[tid], arr[swapIdx]);
                    }
                }
                else{
                    if(arr[tid] < arr[swapIdx]){
                        swap(arr[tid], arr[swapIdx]);
                    }
                }
            }

            __syncthreads();
        }
    }
}


int greatestPowerOfTwoLargerThan(int n)
{
    int k=1;
    while (k<n)
        k=k<<1;
    return k;
}


int main(){
    int* arr= (int*) malloc(Num*sizeof(int));

    time_t t;
    srand((unsigned)time(&t));
    for(int i=0;i<Num;i++){
        arr[i] = rand() % 1000; 
    }


    int* ptr;
    hipError_t err;

    if (Num<=1024){
        int numMax = greatestPowerOfTwoLargerThan(num);

        err = hipMalloc((void**)&ptr, numMax*sizeof(int));
        if (hipSuccess != err){
            cout<<"hipError_t "<<err<<endl;
            return err;
        }

        err = hipMemcpy(ptr, arr, Num*sizeof(int), hipMemcpyHostToDevice);
        if (hipSuccess != err){
            cout<<"hipError_t "<<err<<endl;
            return err;
        }

        littleBinoticSort<<<1, numMax>>>(ptr, Num, numMax);
    }
    else{
        int numMax = greatestPowerOfTwoLargerThan(num);

        err = hipMalloc((void**)&ptr, numMax*sizeof(int));
        if (hipSuccess != err){
            cout<<"hipError_t "<<err<<endl;
            return err;
        }

        err = hipMemcpy(ptr, arr, Num*sizeof(int), hipMemcpyHostToDevice);
        if (hipSuccess != err){
            cout<<"hipError_t "<<err<<endl;
            return err;
        }

        bigBinoticSort<<<1, 1024>>>(ptr, Num, numMax);
    }


    err = hipMemcpy(arr, ptr, Num*sizeof(int), hipMemcpyDeviceToHost);
    if (hipSuccess != err){
        cout<<"hipError_t "<<err<<endl;
        return err;
    }

    for(int i=0;i<Num;i++){
        cout<<arr[i]<<" ";
    }
    cout<<endl;

    err = hipFree(ptr);
    if (hipSuccess != err){
        cout<<"hipError_t "<<err<<endl;
        return err;
    }

    return 0;
}